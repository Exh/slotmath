#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define NUM_REELS 5
#define NUM_LINES 9

typedef struct {
   char symbol;
   float weight;
} symbol_weight_t;

typedef struct {
    char symbol;
    int frequency;
    float value;
} payout_t;

typedef struct {
    char symbol;
    int count;
} symbol_count_t;

char** str_split(char* str, const char* delimiterStr, size_t *count) {
    const char delimiter = delimiterStr[0];
    char** result = NULL;
    *count = 0;

    /* Count how many elements will be extracted. */
    char* tmp = str;
    while (*tmp) {
        if (delimiter == *tmp) {
            *count += 1;
        }
        tmp++;
    }
    if(*count > 0) {
        *count += 1;
        result = (char**) malloc(sizeof(char*) * *count);
        size_t idx = 0;
        char* token = strtok(str, delimiterStr);
        while (token) {
            *(result + idx) = strdup(token);
            idx += 1;
            token = strtok(0, delimiterStr);
        }
    }
    return result;
}
__device__ int isWin(payout_t payout, symbol_weight_t *line) {
    int payout_symbol_count = 0;
    int index = 0;
    for (index = 0; index < NUM_REELS; index++) {
        symbol_weight_t symbol_weight = line[index];
        if(symbol_weight.symbol == payout.symbol) {
            payout_symbol_count += 1;
        }
    }
    if (payout_symbol_count == payout.frequency) {
        return 1;
    }
    return 0;
}

__global__ void calculateExpectedValue(const int num_symbols_per_reel, const int num_payouts, const int total_choices,
									   int *device_possible_lines, symbol_weight_t *device_symbols_weights, payout_t *device_payouts, float *device_expected_values) {
    int a_index = threadIdx.x;
    if (a_index < num_symbols_per_reel) {
		float expected_value = 0.0;
		int b,c,d,e;
		int ai,bi,ci,di,ei;
		for (b = 0; b < num_symbols_per_reel; b++) {
			for (c = 0; c < num_symbols_per_reel; c++) {
				for (d = 0; d < num_symbols_per_reel; d++) {
					for (e = 0; e < num_symbols_per_reel; e++) {
						symbol_weight_t lines[NUM_LINES][NUM_REELS];
						int i = 0;
						for(i = 0; i < NUM_LINES; i++) {
							int possible_line[NUM_REELS];
							int j = 0;
							for(j = 0; j < NUM_REELS; j++) {
								possible_line[j] = device_possible_lines[i*NUM_REELS + j];
							}

							ai = a_index + possible_line[0];
							if (ai == num_symbols_per_reel) {
								ai = 0;
							}
							if (ai < 0) {
								ai = num_symbols_per_reel-1;
							}
							bi = b + possible_line[1];
							if (bi == num_symbols_per_reel) {
								bi = 0;
							}
							if (bi < 0) {
								bi = num_symbols_per_reel-1;
							}
							ci = c + possible_line[2];
							if (ci == num_symbols_per_reel) {
								ci = 0;
							}
							if (ci < 0) {
								ci = num_symbols_per_reel-1;
							}
							di = d + possible_line[3];
							if (di == num_symbols_per_reel) {
								di = 0;
							}
							if (di < 0) {
								di = num_symbols_per_reel-1;
							}
							ei = e + possible_line[4];
							if (ei == num_symbols_per_reel) {
								ei = 0;
							}
							if (ei < 0) {
								ei = num_symbols_per_reel-1;
							}
							lines[i][0] = device_symbols_weights[ai*NUM_REELS];
							lines[i][1] = device_symbols_weights[bi*NUM_REELS + 1];
							lines[i][2] = device_symbols_weights[ci*NUM_REELS + 2];
							lines[i][3] = device_symbols_weights[di*NUM_REELS + 3];
							lines[i][4] = device_symbols_weights[ei*NUM_REELS + 4];
						}
						for(i = 0; i < NUM_LINES; i++) {
							int j = 0;
							for(j = 0; j < num_payouts; j++) {
								if(isWin(device_payouts[j], lines[i])) {
									int k = 0;
									float probability = 1.0;
									for(k = 0; k < NUM_REELS; k++) {
										probability *= lines[0][k].weight;
									}
									probability /= total_choices;
									expected_value += device_payouts[j].value * probability; 
								}
							}
						}
					}
				}
			}
		}
		device_expected_values[a_index] = expected_value;
    }
}

int main(void) {
    
    //get the symbols and weights on each reel
    printf("**Reading symbols and weights file\n");
    FILE *reels_weights_file = fopen("reels_weights_shuffled.csv", "rb");
    if (reels_weights_file == NULL) {
        printf("cannot open reels/weights file\n");
        return 1;
    }
    char line [512];
    int num_symbols_per_reel = 0;
    while (fgets(line, 512, reels_weights_file) != NULL) {
        size_t num_tokens = 0;
        char **tokens = str_split(line, ",", &num_tokens);
        if (num_tokens != NUM_REELS) {
            break;
        }
        num_symbols_per_reel += 1;
    }

    symbol_weight_t *symbols_weights = (symbol_weight_t*) malloc(sizeof(symbol_weight_t) * num_symbols_per_reel * NUM_REELS);
    rewind(reels_weights_file);
    int index = 0;
    while (fgets(line, 512, reels_weights_file) != NULL) {
        size_t num_tokens = 0;
        char **tokens = str_split(line, ",", &num_tokens);
        if (num_tokens != NUM_REELS) {
            break;
        }
        int i = 0;
        for(i = 0; i < num_tokens; i++) {
            size_t num_strs = 0;
            char **strs = str_split(tokens[i], "_", &num_strs);
            if(num_strs == 2) {
                char symbol = strs[0][0];
                float weight = strtof(strs[1], NULL);
                symbol_weight_t symbol_weight = {symbol, weight};
                symbols_weights[index * NUM_REELS + i] = symbol_weight;
            }
        }
        index += 1;
    }
    fclose (reels_weights_file);

    //get the payouts
    printf("**Reading payouts file\n");
    FILE *payouts_file = fopen("payouts.csv", "rb");
    if (payouts_file == NULL) {
        printf("cannot open payouts file\n");
        return 1;
    }
    int num_payouts = 0;
    while (fgets(line, 512, payouts_file) != NULL) {
        num_payouts += 1;
    }

    payout_t *payouts = (payout_t*) malloc(sizeof(payout_t) * num_payouts);
    rewind(payouts_file);
    int payout_index = 0;
    while (fgets(line, 512, payouts_file) != NULL) {
        size_t num_tokens = 0;
        char **tokens = str_split(line, ",", &num_tokens);
        char symbol = tokens[0][0];
        int frequency = atoi(tokens[1]);
        float value = strtof(tokens[2], NULL);
        payout_t payout = {symbol, frequency, value};
        payouts[payout_index] = payout;
        payout_index += 1;
    }
    fclose(payouts_file);

    printf("**Calculating the total symbol weight for each reel\n");
    int reel_weights[NUM_REELS];
    int i = 0;
    for(i = 0; i < NUM_REELS; i++) {
        reel_weights[i] = 0;
    }
    for(i = 0; i < num_symbols_per_reel; i++) {
        int j = 0;
        for(j = 0; j < NUM_REELS; j++) {
        	symbol_weight_t symbol_weight = symbols_weights[i*NUM_REELS + j];
            reel_weights[j] += symbol_weight.weight;
        }
    }

    printf("**Initializing lines\n");
    int possible_lines[NUM_LINES * NUM_REELS] = {
    							 0,0,0,0,0,
                                 -1,-1,-1,-1,-1,
                                 1,1,1,1,1,
                                 -1,0,1,0,-1,
                                 1,0,-1,0,1,
                                 0,-1,-1,-1,0,
                                 0,1,1,1,0,
                                 -1,-1,0,1,1,
                                 1,1,0,-1,-1};

    printf("**Finding expected value\n");
    int total_choices = 1;
    for(i = 0; i < NUM_REELS; i++) {
        total_choices *= reel_weights[i];
    }

    //copy host possible lines to device possible lines
    int *device_possible_lines = NULL;
    size_t size = sizeof(int) * NUM_LINES * NUM_REELS;
    hipError_t error = hipMalloc((void **)&device_possible_lines, size);
    if (error != hipSuccess) {
    	printf("hipMalloc device_possible_lines returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(device_possible_lines, possible_lines, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
    	printf("hipMemcpy device_possible_lines returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    //copy host symbols weights to device symbols weights
    symbol_weight_t *device_symbols_weights = NULL;
    size = num_symbols_per_reel * NUM_REELS * sizeof(symbol_weight_t);
    error = hipMalloc((void **)&device_symbols_weights, size);
    if (error != hipSuccess) {
       	printf("hipMalloc device_symbols_weights returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(device_symbols_weights, symbols_weights, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
    	printf("hipMemcpy device_symbols_weights returned error code %d, line(%d)\n", error, __LINE__);
    	exit(EXIT_FAILURE);
    }

    //copy host payouts to device payouts
    payout_t *device_payouts = NULL;
    size = num_payouts * sizeof(payout_t);
    error = hipMalloc((void**)&device_payouts, size);
    if (error != hipSuccess) {
    	printf("hipMalloc device_payouts returned error code %d, line(%d)\n", error, __LINE__);
    	exit(EXIT_FAILURE);
    }
    error = hipMemcpy(device_payouts, payouts, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
    	printf("hipMemcpy device_payouts returned error code %d, line(%d)\n", error, __LINE__);
    	exit(EXIT_FAILURE);
    }

    //initialize expected value array, this will hold the expected value calculated by each thread
    size = num_symbols_per_reel * sizeof(float);
    float *expected_values = (float*) malloc(size);
    float *device_expected_values = NULL;
    error = hipMalloc((void**)&device_expected_values, size);
    if (error != hipSuccess) {
       	printf("hipMalloc device_expected_values returned error code %d, line(%d)\n", error, __LINE__);
       	exit(EXIT_FAILURE);
    }

    //invoke the device code
    int threadsPerBlock = num_symbols_per_reel;
    int blocksPerGrid = 1;//(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    calculateExpectedValue<<<blocksPerGrid, threadsPerBlock>>>(num_symbols_per_reel, num_payouts, total_choices,
    		device_possible_lines, device_symbols_weights, device_payouts, device_expected_values);
    error = hipGetLastError();
    if (error != hipSuccess) {
    	fprintf(stderr, "Failed to launch calculateExpectedValue kernel (error code %s)!\n", hipGetErrorString(error));
    	exit(EXIT_FAILURE);
    }

    //copy expected values from CUDA device to host memory
    error = hipMemcpy(expected_values, device_expected_values, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess){
    	fprintf(stderr, "Failed to copy vector expected values from device to host (error code %s)!\n", hipGetErrorString(error));
    	exit(EXIT_FAILURE);
    }

    //sum each of the device expected values into one sum
    float expected_value = 0.0;
    for(i = 0; i < num_symbols_per_reel; i++) {
    	expected_value += expected_values[i];
    }
    printf("expected value %f\n", expected_value/NUM_LINES);

    //free host and device memory
    free(symbols_weights);
    free(payouts);
    free(expected_values);
    hipFree(device_possible_lines);
    hipFree(device_symbols_weights);
    hipFree(device_payouts);
    hipFree(device_expected_values);

    hipDeviceReset();

    return 0;
}
